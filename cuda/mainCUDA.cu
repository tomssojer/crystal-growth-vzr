#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include "../constants.h"
#include "modelCUDA.h"
#include "hip/hip_runtime_api.h"

int THREADS_PER_BLOCK;
__device__ bool stopProcessing = false;

__global__ void testGPU()
{
    // printf("Hello world from the GPU!\n");
}

__global__ void stop_sim(Cell *d_cells)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;

    if (x < NUM_CELLS) // ne presežem limite slike
    {
        // Če je ena od sosed celice tipa edge, prekini simulacijo
        if (d_cells[x].type == 1)
        {

            for (int k = 0; k < NUM_NEIGHBORS; k++)
            {
                int sosed = d_cells[x].neighbors[k];
                if (sosed >= 0)
                {
                    if (d_cells[sosed].type == 3)
                    {
                        // return something  to driver function so it stops
                        stopProcessing = true;
                    }
                }
            }
        }
    }
}
__global__ void cell_type(Cell *d_cells, double *stateTemp)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;

    if (x < NUM_CELLS) // ne presežem limite slike
    {
        d_cells[x].state = stateTemp[x];
        if (d_cells[x].state >= 1)
        {
            d_cells[x].type = 0; // turns into ice cell
            for (int i = 0; i < NUM_NEIGHBORS; i++)
            {
                int sosed = d_cells[x].neighbors[i];
                // Preveri, da je valid sosed
                if (sosed >= 0)
                {
                    // Dodeli tip boundary le, če ni frozen ali edge
                    if (d_cells[sosed].type != 0 && d_cells[sosed].type != 3)
                    {
                        d_cells[sosed].type = 1;
                    }
                }
            }
        }
    }
}

__global__ void cell_type_cache(Cell *d_cells, double *stateTemp)
{
    __shared__ Cell cache[16];
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int threadId = threadIdx.x;

    if (x < NUM_CELLS) // ne presežem limite slike
    {
        d_cells[x].state = stateTemp[x];
        if (d_cells[x].state >= 1)
        {
            d_cells[x].type = 0;

            // cache cells after modifying them
            cache[threadId] = d_cells[x];
            __syncthreads();

            for (int i = 0; i < NUM_NEIGHBORS; i++)
            {
                // ko delamo bralne dostope cachiraj
                int sosed = cache[threadId].neighbors[i];
                //  Preveri, da je valid sosed
                if (sosed >= 0)
                {
                    // printf("vel %d \t",blockIdx.x * blockDim.x);
                    if (sosed >= blockIdx.x * blockDim.x && sosed < (blockIdx.x + 1) * blockDim.x)
                    {
                        if (x - sosed == 1)
                        {
                            if (cache[threadId - 1].type == 2)
                                d_cells[sosed].type = 1;
                        }
                        else if (sosed - x == 1)
                        {
                            if (cache[threadId + 1].type == 2)
                                d_cells[sosed].type = 1;
                        }
                        else if (d_cells[sosed].type == 2)
                        {
                            d_cells[sosed].type = 1;
                        }
                    }
                    else if (d_cells[sosed].type == 2)
                    {
                        d_cells[sosed].type = 1;
                    }
                }
            }
        }
    }
}

__global__ void get_states(Cell *d_cells, double *stateTemp, int size)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;

    if (x < size && d_cells[x].type != 3) // ne presežem limite slike
    {

        double state = d_cells[x].state;
        double average = 0.0;
        int *neighbors = d_cells[x].neighbors;
        for (int i = 0; i < NUM_NEIGHBORS; i++)
        {
            int sosed = neighbors[i];
            if (sosed >= 0)
            {
                // Če je type sosednje celice unreceptive ali edge, potem pridobi del od nje
                if (d_cells[sosed].type > 1)
                {
                    average += d_cells[sosed].state;
                }
            }
        }

        average = average / NUM_NEIGHBORS;

        int type = d_cells[x].type;
        if (type < 2)
        {
            state = state + (ALPHA / 2) * average + GAMMA;
        }
        //  unreceptive, edge
        else
        {
            state = state + ALPHA / 2 * (average - state);
        }

        stateTemp[x] = state; // cells[x].state + double((ALPHA/2)) + GAMMA; //state;
    }
}
__global__ void get_states_cache(Cell *d_cells, double *stateTemp, int size)
{
    extern __shared__ Cell cache[];
    int x = threadIdx.x + blockIdx.x * blockDim.x;

    if (x < size)
    {
        int threadId = threadIdx.x;

        cache[threadId] = d_cells[x];
        __syncthreads();

        if (cache[threadId].type != 3) // ne presežem limite slike
        {
            double state = cache[threadId].state;
            double average = 0.0;

            for (int i = 0; i < NUM_NEIGHBORS; i++)
            {
                int sosed = cache[threadId].neighbors[i];
                Cell neighbor_cell;

                if (sosed >= 0)
                {
                    if (sosed >= blockIdx.x * blockDim.x && sosed < (blockIdx.x + 1) * blockDim.x)
                    {
                        if (x - sosed == 1)
                            neighbor_cell = cache[threadId - 1];
                        else if (sosed - x == 1)
                            neighbor_cell = cache[threadId + 1];
                        else
                            neighbor_cell = d_cells[sosed];
                    }
                    else
                        neighbor_cell = d_cells[sosed];

                    // Če je type sosednje celice unreceptive ali edge, potem pridobi del od nje
                    if (neighbor_cell.type > 1)
                        average += neighbor_cell.state;
                }
            }

            average = average / NUM_NEIGHBORS;

            int type = cache[threadId].type;
            if (type < 2)
            {
                state = state + (ALPHA / 2) * average + GAMMA;
            }
            //  unreceptive, edge
            else
            {
                state = state + ALPHA / 2 * (average - state);
            }

            stateTemp[x] = state; // cells[x].state + double((ALPHA/2)) + GAMMA; //state;
        }
    }
}

void parallel_cuda(Cell *d_cells, Cell *cells, int blocksize, int cache_flag)
{
    double *d_stateTemp;
    checkCudaErrors(hipMalloc((void **)&d_stateTemp, NUM_CELLS * sizeof(double)));
    int numBlocks = (NUM_CELLS + blocksize - 1) / blocksize;

    for (int i = 0; i < STEPS; i++) // iteracije, oz stanja po casu
    {
        bool stopFlagValue;
        // update states of board
        if (cache_flag)
            get_states_cache<<<numBlocks, blocksize, THREADS_PER_BLOCK>>>(d_cells, d_stateTemp, NUM_CELLS);
        else
            get_states<<<numBlocks, blocksize>>>(d_cells, d_stateTemp, NUM_CELLS);

        hipDeviceSynchronize();
        cell_type<<<numBlocks, blocksize>>>(d_cells, d_stateTemp);
        hipDeviceSynchronize();

        stop_sim<<<numBlocks, blocksize>>>(d_cells);
        hipDeviceSynchronize();
        hipMemcpyFromSymbol(&stopFlagValue, HIP_SYMBOL(stopProcessing), sizeof(bool));

        if (stopFlagValue)
        {
            printf("Breaking %d\n", i);
            i = STEPS;
            break;
        }
    }

    checkCudaErrors(hipMemcpy(cells, d_cells, NUM_CELLS * sizeof(Cell), hipMemcpyDeviceToHost));
    getLastCudaError("printGPU() execution failed\n");
    hipFree(d_stateTemp);
}
void check_CUDA() // function to copy into GPU memory
{
    int deviceCount;
    printf("Hello world from the CPU!\n");

    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0)
    {
        printf("No CUDA devices found\n");
    }

    testGPU<<<1, 1>>>(); // gred size block size
    hipDeviceSynchronize();
}
void run_CUDA(Cell *cells, int blocksize, int cache_flag)
{
    // Allocate memory on GPU
    Cell *d_cells;
    checkCudaErrors(hipMalloc((void **)&d_cells, NUM_CELLS * sizeof(Cell)));
    checkCudaErrors(hipMemcpy(d_cells, cells, NUM_CELLS * sizeof(Cell), hipMemcpyHostToDevice));
    hipDeviceSynchronize();
    getLastCudaError("printGPU() execution failed\n");

    parallel_cuda(d_cells, cells, blocksize, cache_flag);
    // Free memory on GPU
    hipFree(d_cells);
}
void get_cacheSize(int size)
{
    int deviceId;
    hipGetDevice(&deviceId);

    hipDeviceProp_t deviceProps;
    hipGetDeviceProperties(&deviceProps, deviceId);

    int sharedMemorySize = deviceProps.sharedMemPerBlock;

    printf("Max shared memory size per block: %d bytes\n", sharedMemorySize / size);
}
int main(int argc, char *argv[])
{

    if (argc < 3)
    {
        printf("Not enough arguments!\n");
        return 1;
    }

    int blocksize = atoi(argv[1]);
    int cache_flag = atoi(argv[2]);

    THREADS_PER_BLOCK = blocksize;

    // // ------------- Začetek inicializacije ------------- //
    // Definicija arraya s structi
    Cell *cells = (Cell *)malloc(NUM_CELLS * sizeof(*cells));
    // get_cacheSize(sizeof(Cell));
    init_grid(cells);
    // Določi začetno vrednost glede na tip celice
    init_state(cells);

    // ------------- Konec inicializacije ------------- //

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // draw_board(cells);
    run_CUDA(cells, blocksize, cache_flag);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Elapsed time: %0.3f seconds \n", milliseconds / 1000);

    // draw_board(cells);

    // Free allocated memory
    free(cells);

    return 0;
}