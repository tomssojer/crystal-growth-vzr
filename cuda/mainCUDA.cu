#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include "../constants.h"
#include "modelCUDA.h"
#include "hip/hip_runtime_api.h"
#define THREADS_PER_BLOCK 128

__device__ bool stopProcessing = false;

__global__ void testGPU()
{
    // printf("Hello world from the GPU!\n");
}

__global__ void stop_sim(Cell *d_cells)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;

    if (x < NUM_CELLS) // ne presežem limite slike
    {
        // Če je ena od sosed celice tipa edge, prekini simulacijo
        if (d_cells[x].type == 1)
        {

            for (int k = 0; k < NUM_NEIGHBORS; k++)
            {
                int sosed = d_cells[x].neighbors[k];
                if (sosed >= 0)
                {
                    if (d_cells[sosed].type == 3)
                    {
                        // return something  to driver function so it stops
                        stopProcessing = true;
                    }
                }
            }
        }
    }
}
__global__ void cell_type(Cell *d_cells, double *stateTemp)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;

    if (x < NUM_CELLS) // ne presežem limite slike
    {
        d_cells[x].state = stateTemp[x];
        if (d_cells[x].state >= 1)
        {
            d_cells[x].type = 0; // turns into ice cell
            for (int i = 0; i < NUM_NEIGHBORS; i++)
            {
                int sosed = d_cells[x].neighbors[i];
                // Preveri, da je valid sosed
                if (sosed >= 0)
                {
                    // Dodeli tip boundary le, če ni frozen ali edge
                    if (d_cells[sosed].type != 0 && d_cells[sosed].type != 3)
                    {
                        d_cells[sosed].type = 1;
                    }
                }
            }
        }
    }
}
__global__ void get_states(Cell *d_cells, double *stateTemp, int size)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;

    if (x < size && d_cells[x].type != 3) // ne presežem limite slike
    {

        double state = d_cells[x].state;
        double average = 0.0;
        int *neighbors = d_cells[x].neighbors;
        for (int i = 0; i < NUM_NEIGHBORS; i++)
        {

            int sosed = neighbors[i];
            // printf("sosed: %d \t ||",sosed);
            if (sosed >= 0)
            {
                // Če je type sosednje celice unreceptive ali edge, potem pridobi del od nje
                if (d_cells[sosed].type > 1)
                {
                    average += d_cells[sosed].state;
                    // printf("x %d je: %d => %f \n",x,sosed,d_cells[sosed].state);
                }
            }
        }

        average = average / NUM_NEIGHBORS;

        int type = d_cells[x].type;
        if (type < 2)
        {
            state = state + (ALPHA / 2) * average + GAMMA;
        }
        //  unreceptive, edge
        else
        {
            state = state + ALPHA / 2 * (average - state);
        }

        stateTemp[x] = state; // cells[x].state + double((ALPHA/2)) + GAMMA; //state;
    }
}
__global__ void get_states_cache(Cell *d_cells, double *stateTemp, int size)
{
     __shared__ Cell cache[7*THREADS_PER_BLOCK];
    //__shared__ Cell cache[THREADS_PER_BLOCK];
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    if (x < size)  {
        int threadId = threadIdx.x;
       
        cache[threadId] = d_cells[x];
        __syncthreads();

        if (cache[threadId].type != 3) // ne presežem limite slike
        {
            double state = cache[threadId].state;
            double average = 0.0;
            
            for (int i = 0; i < NUM_NEIGHBORS; i++)
            {
                int sosed = cache[threadId].neighbors[i];

                // printf("sosed: %d \t ||",sosed);
                if (sosed >= 0)
                {
                    cache[(i+1)*THREADS_PER_BLOCK + threadId] = d_cells[sosed];
                    // Če je type sosednje celice unreceptive ali edge, potem pridobi del od nje
                    if (cache[(i+1)*THREADS_PER_BLOCK+ threadId].type > 1)
                    {
                        average += cache[(i+1)*THREADS_PER_BLOCK + threadId].state;
                            //printf("avg %d \t",avrage);
                        // printf("x %d je: %d => %f \n",x,sosed,d_cells[sosed].state);
                    }
                }
            }
            average = average / NUM_NEIGHBORS;

            int type =   cache[threadId].type;
            if (type < 2)
            {
                state = state + (ALPHA / 2) * average + GAMMA;
            }
            //  unreceptive, edge
            else
            {
                state = state + ALPHA / 2 * (average - state);
            }

            stateTemp[x] = state; // cells[x].state + double((ALPHA/2)) + GAMMA; //state;
        }
     }
}

void parallel_cuda(Cell *d_cells, Cell *cells, int blockSize)
{
    double *d_stateTemp;
    checkCudaErrors(hipMalloc((void **)&d_stateTemp, NUM_CELLS * sizeof(double)));
    int numBlocks = (NUM_CELLS + blockSize - 1) / blockSize;

    for (int i = 0; i < STEPS; i++) // iteracije, oz stanja po casu
    {
        bool stopFlagValue;
        // update states of board
        get_states_cache<<<numBlocks, blockSize>>>(d_cells, d_stateTemp, NUM_CELLS);
        hipDeviceSynchronize();
        cell_type<<<numBlocks, blockSize>>>(d_cells, d_stateTemp);
        hipDeviceSynchronize();

        stop_sim<<<numBlocks, blockSize>>>(d_cells);
        hipDeviceSynchronize();
        hipMemcpyFromSymbol(&stopFlagValue, HIP_SYMBOL(stopProcessing), sizeof(bool));

        if (stopFlagValue)
        {
            printf("breking %d\n", i);
            i = STEPS;
            break;
        }

        // if (i % STEPS_TO_DRAW == 0)
        // {
        //      checkCudaErrors(hipMemcpy(cells, d_cells, NUM_CELLS * sizeof(Cell), hipMemcpyDeviceToHost));
        //     // printf("Step number: %d\n", i);
        //     draw_board(cells);
        //     // write_to_file(cells, file);
        // }
        // printf("Step: %d ----------------------------------------------------------\n", i);
        // draw_board(cells);
    }
    checkCudaErrors(hipMemcpy(cells, d_cells, NUM_CELLS * sizeof(Cell), hipMemcpyDeviceToHost));
    getLastCudaError("printGPU() execution failed\n");
    hipFree(d_stateTemp);
}
void check_CUDA() // function to copy into GPU memory
{
    int deviceCount;
    printf("Hello world from the CPU!\n");

    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0)
    {
        printf("No CUDA devices found\n");
    }

    testGPU<<<1, 1>>>(); // gred size block size
    hipDeviceSynchronize();
}
void run_CUDA(Cell *cells, int blocksize)
{
    // Allocate memory on GPU
    Cell *d_cells;
    checkCudaErrors(hipMalloc((void **)&d_cells, NUM_CELLS * sizeof(Cell)));
    checkCudaErrors(hipMemcpy(d_cells, cells, NUM_CELLS * sizeof(Cell), hipMemcpyHostToDevice));
    hipDeviceSynchronize();
    getLastCudaError("printGPU() execution failed\n");

    parallel_cuda(d_cells, cells, blocksize);
    // Free memory on GPU
    hipFree(d_cells);
}

int main(int argc, char *argv[])
{

    // if (argc < 2)
    // {
    //     printf("Not enough arguments!\n");
    //     return 1;
    // }

    // int blocksize = atoi(argv[1]);

    // // ------------- Začetek inicializacije ------------- //
    // Definicija arraya s structi
    Cell *cells = (Cell *)malloc(NUM_CELLS * sizeof(*cells));

    // Dodaj sosede in indekse v struct
    init_grid(cells);

    // Določi začetno vrednost glede na tip celice
    init_state(cells);

    // ------------- Konec inicializacije ------------- //

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // draw_board(cells);
    run_CUDA(cells, THREADS_PER_BLOCK);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Elapsed time: %0.3f seconds \n", milliseconds / 1000);

    //draw_board(cells);

    // Free allocated memory
    free(cells);

    return 0;
}