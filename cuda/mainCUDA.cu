#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include "../constants.h"
#include "modelCUDA.h"
#include "hip/hip_runtime_api.h"

// compile
// nvcc mainCUDA.cu -O2  -o mainCUDA

// 1. Začni z eno frozen celico, okoli nje so boundary
// 2. Za vse celice, ki so boundary in unreceptive poteka difuzija
// 3. Za vse celice, ki so frozen in boundary poteka konvekcija
// 3.a Upoštevaj, da le sosede, ki so edge ali unreceptive sharajo vodo
// 4. Preveri, če ima celica state >= 1 -> nastavi na frozen, njene sosede na boundary
// 5. Preveri, če je boundary celica soseda z edge celico, prekini simulacijo

__device__ bool stopProcessing = false;

__global__ void testGPU()
{
    // printf("Hello world from the GPU!\n");
}

__global__ void stop_sim(Cell *d_cells)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;

    if (x < NUM_CELLS) // ne presežem limite slike
    {
        // Če je ena od sosed celice tipa edge, prekini simulacijo
        if (d_cells[x].type == 1)
        {

            for (int k = 0; k < NUM_NEIGHBORS; k++)
            {
                if (d_cells[x].neighbors[k] == 3)
                {
                    // printf("break %d\n", x);
                    stopProcessing = true;
                    // return something  to driver function so it stops
                }
            }
        }
    }
}
__global__ void cell_type(Cell *d_cells, double *stateTemp)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;

    if (x < NUM_CELLS) // ne presežem limite slike
    {
        d_cells[x].state = stateTemp[x];
        if (d_cells[x].state >= 1)
        {
            d_cells[x].type = 0; // turns into ice cell
            for (int i = 0; i < NUM_NEIGHBORS; i++)
            {
                int sosed = d_cells[x].neighbors[i];
                // Preveri, da je valid sosed
                if (sosed >= 0)
                {
                    // Dodeli tip boundary le, če ni frozen ali edge
                    if (d_cells[sosed].type != 0 && d_cells[sosed].type != 3)
                    {
                        d_cells[sosed].type = 1;
                        // for (int k = 0; k < NUM_NEIGHBORS; k++)
                        // {
                        //     if (d_cells[x].neighbors[k] == 3)
                        //     {
                        //         printf("break %d\n", x);
                        //         stopProcessing = true;
                        //         // return something  to driver function so it stops
                        //     }
                        // }
                    }
                }
            }
        }
    }
}

__global__ void get_states(Cell *d_cells, double *stateTemp, int size)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;

    if (x < size && d_cells[x].type != 3) // ne presežem limite slike
    {

        double state = d_cells[x].state;
        double average = 0.0;
        int *neighbors = d_cells[x].neighbors;
        for (int i = 0; i < NUM_NEIGHBORS; i++)
        {

            int sosed = neighbors[i];
            // printf("sosed: %d \t ||",sosed);
            if (sosed >= 0)
            {
                // Če je type sosednje celice unreceptive ali edge, potem pridobi del od nje
                if (d_cells[sosed].type > 1)
                {
                    average += d_cells[sosed].state;
                    // printf("x %d je: %d => %f \n",x,sosed,d_cells[sosed].state);
                }
            }
        }

        average = average / NUM_NEIGHBORS;

        int type = d_cells[x].type;
        if (type < 2)
        {
            state = state + (ALPHA / 2) * average + GAMMA;
        }
        //  unreceptive, edge
        else
        {
            state = state + ALPHA / 2 * (average - state);
        }

        stateTemp[x] = state; // cells[x].state + double((ALPHA/2)) + GAMMA; //state;
    }
}

void parallel_cuda(Cell *d_cells, Cell *cells)
{
    double *d_stateTemp;
    checkCudaErrors(hipMalloc((void **)&d_stateTemp, NUM_CELLS * sizeof(double)));
    int blockSize = 512;
    int numBlocks = (NUM_CELLS + blockSize - 1) / blockSize;

    for (int i = 0; i < STEPS; i++) // iteracije, oz stanja po casu
    {
        bool stopFlagValue;
        // update states of board
        get_states<<<numBlocks, blockSize>>>(d_cells, d_stateTemp, NUM_CELLS);
        hipDeviceSynchronize();
        cell_type<<<numBlocks, blockSize>>>(d_cells, d_stateTemp);
        hipDeviceSynchronize();

        stop_sim<<<numBlocks, blockSize>>>(d_cells);
        hipDeviceSynchronize();
        hipMemcpyFromSymbol(&stopFlagValue, HIP_SYMBOL(stopProcessing), sizeof(bool));

        if (stopFlagValue)
        {
            // printf("\nSTEP breking %d\n", i);
            i = STEPS;
            break;
        }

        // if (i % STEPS_TO_DRAW == 0)
        // {
        //      checkCudaErrors(hipMemcpy(cells, d_cells, NUM_CELLS * sizeof(Cell), hipMemcpyDeviceToHost));
        //     // printf("Step number: %d\n", i);
        //     draw_board(cells);
        //     // write_to_file(cells, file);
        // }
        // printf("Step: %d ----------------------------------------------------------\n", i);
        // draw_board(cells);
    }
    checkCudaErrors(hipMemcpy(cells, d_cells, NUM_CELLS * sizeof(Cell), hipMemcpyDeviceToHost));
    getLastCudaError("printGPU() execution failed\n");
    hipFree(d_stateTemp);
}
void check_CUDA() // function to copy into GPU memory
{
    int deviceCount;
    printf("Hello world from the CPU!\n");

    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0)
    {
        printf("No CUDA devices found\n");
    }

    testGPU<<<1, 1>>>(); // gred size block size
    hipDeviceSynchronize();
}
void run_CUDA(Cell *cells)
{
    // Allocate memory on GPU
    Cell *d_cells;
    checkCudaErrors(hipMalloc((void **)&d_cells, NUM_CELLS * sizeof(Cell)));
    checkCudaErrors(hipMemcpy(d_cells, cells, NUM_CELLS * sizeof(Cell), hipMemcpyHostToDevice));
    hipDeviceSynchronize();
    getLastCudaError("printGPU() execution failed\n");

    parallel_cuda(d_cells, cells);
    // Free memory on GPU
    hipFree(d_cells);
}

int main(int argc, char *argv[])
{
    // // ------------- Začetek inicializacije ------------- //
    // printHexagon(ROWS); //

    // Definicija arraya s structi
    Cell *cells = (Cell *)malloc(NUM_CELLS * sizeof(*cells));

    // Dodaj sosede in indekse v struct
    init_grid(cells);

    // Določi začetno vrednost glede na tip celice
    init_state(cells);

    // ------------- Konec inicializacije ------------- //

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // draw_board(cells);

    // check_CUDA();

    run_CUDA(cells);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Elapsed time: %0.3f seconds \n", milliseconds / 1000);

    // draw_board(cells);

    // Free allocated memory
    free(cells);

    return 0;
}