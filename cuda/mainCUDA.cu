#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include "../constants.h"
#include "modelCUDA.h"
#include "hip/hip_runtime_api.h"

// compile
// nvcc mainCUDA.cu -O2  -o mainCUDA

// 1. Začni z eno frozen celico, okoli nje so boundary
// 2. Za vse celice, ki so boundary in unreceptive poteka difuzija
// 3. Za vse celice, ki so frozen in boundary poteka konvekcija
// 3.a Upoštevaj, da le sosede, ki so edge ali unreceptive sharajo vodo
// 4. Preveri, če ima celica state >= 1 -> nastavi na frozen, njene sosede na boundary
// 5. Preveri, če je boundary celica soseda z edge celico, prekini simulacijo
__global__ void testGPU()
{
    printf("Hello world from the GPU!\n");
}

__global__ void get_states(Cell *d_cells,Cell *h_cells,double *stateTemp, int size)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;

    if (x < size && d_cells[x].type!=3) // ne presežem limite slike
    {

        double state = d_cells[x].state;
        double average = 0.0;
        int* neighbors = d_cells[x].neighbors;
        for (int i = 0; i < 6; i++)
        {   
            
            int sosed= neighbors[i];
            //printf("sosed: %d \t ||",sosed);
            if (sosed >= 0)
            {
                // Če je type sosednje celice unreceptive ali edge, potem pridobi del od nje
                if (d_cells[sosed].type > 1)
                {
                    average += d_cells[sosed].state;
                    //printf("x %d je: %d => %f \n",x,sosed,d_cells[sosed].state);
                }
            }
        }

        average = average/ 6;

        int type = d_cells[x].type;
        if (type < 2)
        {
            state = state + (ALPHA / 2) * average + GAMMA;
        }
        //  unreceptive, edge
        else
        {
            state = state + ALPHA / 2 * (average - state);
        }
     
        
        stateTemp[x] =  state;  // cells[x].state + double((ALPHA/2)) + GAMMA; //state;
    }
}


void parallel_cuda(Cell *d_cells,Cell *cells)
{
    //printf("x: delaaaa %d\t",NUM_CELLS);
    double *stateT= (double*)calloc(NUM_CELLS, sizeof(double));
       
    double *d_stateTemp;
    checkCudaErrors(hipMalloc((void **)&d_stateTemp, NUM_CELLS * sizeof(double)));
    //hipMemset(d_stateTemp, 0, NUM_CELLS * sizeof(double));
    //init d_stateTemp on GPU
    
    int blockSize = 128;
    int numBlocks = (NUM_CELLS + blockSize - 1) / blockSize;
    for (int i = 0; i < STEPS; i++) // iteracije, oz stanja po casu
    {
        //hipMemcpy(d_cells, cells, NUM_CELLS * sizeof(Cell), hipMemcpyHostToDevice);
        //hipDeviceSynchronize();
        // update states of board
        get_states<<<numBlocks, blockSize>>>(d_cells,cells, d_stateTemp, NUM_CELLS);
        hipDeviceSynchronize(); 
        checkCudaErrors(hipMemcpy(stateT, d_stateTemp, NUM_CELLS * sizeof(double), hipMemcpyDeviceToHost));

        //Cell *cells2 = (Cell *)malloc(NUM_CELLS * sizeof(*cells2));
        checkCudaErrors(hipMemcpy(cells, d_cells, NUM_CELLS * sizeof(Cell), hipMemcpyDeviceToHost));
        hipDeviceSynchronize();
        //printf("tu\n");
        // for (int k = 0; k < NUM_CELLS; k++)
        // {
        //     //stateT[k]=0;
        //   printf("average: %lf\n",stateT[k]);
        // }
        // for(int i=0; i< NUM_CELLS;i++ ) {
        //     for(int j=0;j<6;j++) {
        //         printf("%d\n",cells2[i].neighbors[j]);
        //     }
        // }
        //free(cells2);
            
        // }
        for (int j = 0; j < NUM_CELLS; j++) // sedaj posodobi tipe celic
        {
            cells[j].state = stateT[j];
            if (cells[j].state >= 1)
            {
                cells[j].type = 0; // turns into ice cell
                set_type_boundary(cells, cells[j].neighbors);
            }
        }

        // Če je ena od sosed celice tipa edge, prekini simulacijo
        for (int j = 0; j < NUM_CELLS; j++)
        {
            for (int k = 0; k < NUM_NEIGHBORS; k++)
            {
                if (cells[j].type == 1 && cells[j].neighbors[k] == 3)
                {
                    printf("break %d\n", i);
                    i = STEPS;
                    j = NUM_CELLS;
                    break;
                }
            }
        }

        // printf("Step: %d ----------------------------------------------------------\n", i);
        //draw_board(cells);
        checkCudaErrors(hipMemcpy(d_cells, cells, NUM_CELLS * sizeof(Cell), hipMemcpyHostToDevice));
        
    }
    getLastCudaError("printGPU() execution failed\n");
    free(stateT);
    hipFree(d_stateTemp);
}
void check_CUDA() // function to copy into GPU memory
{
    int deviceCount;
    printf("Hello world from the CPU!\n");

    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0)
    {
        printf("No CUDA devices found\n");
    }

    testGPU<<<1, 1>>>(); // gred size block size
    hipDeviceSynchronize();
}
void run_CUDA(Cell *cells)
{
    // Allocate memory on GPU
    Cell *d_cells;
    checkCudaErrors(hipMalloc((void **)&d_cells, NUM_CELLS * sizeof(Cell)));
    checkCudaErrors(hipMemcpy(d_cells, cells, NUM_CELLS * sizeof(Cell), hipMemcpyHostToDevice));

    hipDeviceSynchronize();

 
     getLastCudaError("printGPU() execution failed\n");
    // for(int i=0;i<NUM_CELLS;i++ ) {
    //     checkCudaErrors(hipMemcpy(d_cells[i].neighbors, cells[i].neighbors, 6 * sizeof(int), hipMemcpyHostToDevice));
    // }
    //printf("se dela");
    // Run kernel
    parallel_cuda(d_cells,cells);
    printf(" se dela 2\n");
    // Copy data from GPU to CPU
       //hipDeviceSynchronize();
   //Cell *cells2 = (Cell *)malloc(2*NUM_CELLS * sizeof(*cells));

    //checkCudaErrors(hipMemcpy(cells, d_cells, NUM_CELLS * sizeof(Cell), hipMemcpyDeviceToHost));

    // Free memory on GPU
    hipFree(d_cells);
  
}

int main(int argc, char *argv[])
{
    // // ------------- Začetek inicializacije ------------- //
    // printHexagon(ROWS); //

    // Definicija arraya s structi
    Cell *cells = (Cell *)malloc(NUM_CELLS * sizeof(*cells));
      
    // Dodaj sosede in indekse v struct
    init_grid(cells);

    // Določi začetno vrednost glede na tip celice
    init_state(cells);

    // ------------- Konec inicializacije ------------- //

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    draw_board(cells);

    //check_CUDA();
    printf("Starting simulation...\n");
    run_CUDA(cells);

    // serial(cells);
    draw_board(cells);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Elapsed time: %0.3f milliseconds \n", milliseconds);

    // Free allocated memory
    // for (int i = 0; i < NUM_CELLS; i++)
    // {
    //     free(cells[i].neighbors);
    // }
    free(cells);

    return 0;
}