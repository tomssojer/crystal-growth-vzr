#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "constants.h"
#include "modelCUDA.h"

// compile
// nvcc mainCUDA.cu -O2  -o mainCUDA

// 1. Začni z eno frozen celico, okoli nje so boundary
// 2. Za vse celice, ki so boundary in unreceptive poteka difuzija
// 3. Za vse celice, ki so frozen in boundary poteka konvekcija
// 3.a Upoštevaj, da le sosede, ki so edge ali unreceptive sharajo vodo
// 4. Preveri, če ima celica state >= 1 -> nastavi na frozen, njene sosede na boundary
// 5. Preveri, če je boundary celica soseda z edge celico, prekini simulacijo
__global__ void testGPU()
{
    printf("Hello world from the GPU!\n");
}

__global__ void get_states(Cell *cells, double *stateTemp, int size)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;

    if (x < size && cells[x].type != 3) // ne presežem limite slike
    {
        // average_state(cells[j].neighbors, cells);
    }
}

void printTab(int **tab, int j, int mappIdx)
{
    for (int i = 0; i < 6; i++)
    {
        printf("%d->%d |(%2d,%2d) ", j, mappIdx, tab[i][0], tab[i][1]); // x y
    }
}

void printmapped(int **tab, int j, int x, int *mapp)
{
    for (int i = 0; i < 6; i++)
    {
        printf("[%d,%d]  (%2d,%2d)->%d |", j, x, tab[i][0], tab[i][1], mapp[i]); // x y
    }
}

void printStructs(Cell *cells)
{
    for (int i = 0; i < ROWS * COLUMNS; i++)
    {
        printf("id:%d,\ttype: %d,\tstate: %lf,\tneighbors: ", cells[i].id, cells[i].type, cells[i].state);

        for (int j = 0; j < NUM_NEIGHBORS; j++)
            printf("%d, ", cells[i].neighbors[j]);

        printf("\n");
    }
}

void parallel_cuda(Cell *cells)
{
    float average = 0;
    double *g_stateTemp;
    hipMalloc((void **)&g_stateTemp, NUM_CELLS * sizeof(double));
    // init g_stateTemp on GPU

    for (int i = 0; i < STEPS; i++) // iteracije, oz stanja po casu
    {
        for (int j = 0; j < NUM_CELLS; j++) // posodobi vsa stanja - difuzija, konvekcija
        {
            // We deal with one cell at the time, do not deal with edge type
            if (cells[j].type != 3)
            {
                // Calculate average state of neighbors, needs current cell's neighbours and pointer to all cells

                // average = average_state(cells[j].neighbors, cells);

                // stateTemp[j] = change_state(cells[j].type, cells[j].state, average);
                //  cells[j].state = change_state(cells[j].type, cells[j].state, average);
            }
        }

        for (int j = 0; j < NUM_CELLS; j++) // sedaj posodobi tipe celic
        {
            cells[j].state = stateTemp[j];
            if (cells[j].state >= 1)
            {
                cells[j].type = 0; // turns into ice cell
                set_type_boundary(cells, cells[j].neighbors);
            }
        }

        // Če je ena od sosed celice tipa edge, prekini simulacijo
        for (int j = 0; j < NUM_CELLS; j++)
        {
            for (int k = 0; k < NUM_NEIGHBORS; k++)
            {
                if (cells[j].type == 1 && cells[j].neighbors[k] == 3)
                {
                    printf("break %d\n", i);
                    i = STEPS;
                    j = NUM_CELLS;
                    break;
                }
            }
        }

        // printf("Step: %d ----------------------------------------------------------\n", i);

        // for (int k = 0; k < NUM_CELLS; k++)
        // {
        //     if (cells[k].type == 0 || cells[k].type == 1)
        //         printf("id: %d,\ttype: %d,\tstate: %lf\n", k, cells[k].type, cells[k].state);
        // }
        // printf("\n");

        if (i % 20 == 0)
            draw_board(cells);
    }
    free(stateTemp);
}
void check_CUDA() // function to copy into GPU memory
{
    int deviceCount;
    printf("Hello world from the CPU!\n");

    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0)
    {
        printf("No CUDA devices found\n");
    }

    testGPU<<<1, 1>>>(); // gred size block size
    hipDeviceSynchronize();
}
void run_CUDA(Cell *cells)
{
    // Allocate memory on GPU
    Cell *d_cells;
    hipMalloc((void **)&d_cells, NUM_CELLS * sizeof(Cell));

    // Copy data from CPU to GPU
    hipMemcpy(d_cells, cells, NUM_CELLS * sizeof(Cell), hipMemcpyHostToDevice);

    // Run kernel
    parallel_cuda(d_cells);

    // Copy data from GPU to CPU
    hipMemcpy(cells, d_cells, NUM_CELLS * sizeof(Cell), hipMemcpyDeviceToHost);

    // Free memory on GPU
    hipFree(d_cells);
}

int main(int argc, char *argv[])
{
    // // ------------- Začetek inicializacije ------------- //
    // printHexagon(ROWS); //

    // Definicija arraya s structi
    Cell *cells = (Cell *)malloc(NUM_CELLS * sizeof(*cells));

    // Dodaj sosede in indekse v struct
    init_grid(cells);

    // Določi začetno vrednost glede na tip celice
    init_state(cells);

    // ------------- Konec inicializacije ------------- //

    clock_t start_time, end_time;
    start_time = clock();
    // draw_board(cells);
    check_CUDA();

    // serial(cells);
    // draw_board(cells);

    end_time = clock();
    printf("Time elapsed: %.3lf seconds\n", (double)(end_time - start_time) / CLOCKS_PER_SEC);

    // Free allocated memory
    for (int i = 0; i < NUM_CELLS; i++)
    {
        free(cells[i].neighbors);
    }
    free(cells);

    return 0;
}